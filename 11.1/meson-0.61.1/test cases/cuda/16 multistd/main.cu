#include <hip/hip_runtime.h>
#include <iostream>

auto cuda_devices(void) {
    int result = 0;
    hipGetDeviceCount(&result);
    return result;
}

int do_cuda_stuff();

int main(void) {
    int n = cuda_devices();
    if (n == 0) {
        std::cout << "No Cuda hardware found. Exiting.\n";
        return 0;
    }

    std::cout << "Found " << n << "Cuda devices.\n";
    return do_cuda_stuff();
}
